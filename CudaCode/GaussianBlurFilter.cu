#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "GaussianBlurFilter.h"

double **GBFilter::getGaussian(int height, int width, double sigma)
{
    double **filter;
    double sum = 0.0;
    int i, j;

    filter = new double *[height];
    for (int i = 0; i < height; i++)
        filter[i] = new double[width];

    for (i = 0; i < height; i++)
        for (j = 0; j < width; j++)
        {
            filter[i][j] = exp(-(i * i + j * j) / (2 * sigma * sigma)) / (2 * M_PI * sigma * sigma);
            sum += filter[i][j];
        }

    for (i = 0; i < height; i++)
        for (j = 0; j < width; j++)
            filter[i][j] /= sum;

    return filter;
}

double ***GBFilter::applyFilter(double ***image, double **filter, int width, int height, int filterWidth, int filterHeight)
{

    int newImageHeight = height - filterHeight + 1;
    int newImageWidth = width - filterWidth + 1;
    int d, i, j, h, w;

    double ***newImage;
    newImage = new double **[3];
    for (int i = 0; i < 3; i++)
    {
        newImage[i] = new double *[width];
        for (int j = 0; j < width; j++)
            newImage[i][j] = new double[height];
    }

    for (d = 0; d < 3; d++)
        for (i = 0; i < newImageWidth; i++)
            for (j = 0; j < newImageHeight; j++)
                for (h = i; h < i + filterWidth; h++)
                    for (w = j; w < j + filterHeight; w++)
                        newImage[d][i][j] += filter[h - i][w - j] * image[d][h][w];

    return newImage;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "GaussianBlurFilter.h"

__global__ void cudaKernel(int *n, int limit)
{

}


double **GBFilter::getGaussian(int height, int width, double sigma)
{
    double **filter;
    double sum = 0.0;
    int i, j;

    filter = new double *[height];
    for (i = 0; i < height; i++)
    {
        filter[i] = new double[width];
    }

    for (i = 0; i < height; i++)
    {
        for (j = 0; j < width; j++)
        {
            filter[i][j] = exp(-(i * i + j * j) / (2 * sigma * sigma)) / (2 * M_PI * sigma * sigma);
            sum += filter[i][j];
        }
    }

    for (i = 0; i < height; i++)
    {
        for (j = 0; j < width; j++)
        {
            filter[i][j] /= sum;
        }
    }

    return filter;
}

double **GBFilter::applyFilter(double **image, double **filter, int width, int height)
{
    int filterHeight = 5;
    int filterWidth = 5;
    int newImageHeight = height - filterHeight + 1;
    int newImageWidth = width - filterWidth + 1;
    int i, j, h, w;

    double **newImage;
    newImage = new double *[height];
    {
        for (i = 0; i < height; i++)
        {
            newImage[i] = new double[width];
        }

        for (i = 0; i < newImageHeight; i++)
        {
            for (j = 0; j < newImageWidth; j++)
            {
                for (h = i; h < i + filterHeight; h++)
                {
                    for (w = j; w < j + filterWidth; w++)
                    {
                        newImage[i][j] += filter[h - i][w - j] * image[h][w];
                    }
                }
            }
        }
    }

    return newImage;
}

